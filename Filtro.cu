#include "Prototipos.h"
#include "Prototipos_Alumno.h"

int main(int argc, char* argv[]) 
{
   unsigned char  *Head=NULL;
   char           *Entrada=NULL, *Salida=NULL;
   unsigned int   DimX, DimY, HeadSize, ThBlk;
   double         *Image=NULL;
   int            ndev, Repet, i;
   float          time;
   hipEvent_t    start, stop;
   
   if (argc != 5) {
      printf("Usage: %s <Image input file> <Image output file> <times> <hilos por bloque>\n", argv[0]);
      return -1;
   }

   Entrada = strdup(argv[1]);
   Salida  = strdup(argv[2]);
   Repet   = atoi(argv[3]);
   ThBlk   = atoi(argv[4]);

   CUDAERR(hipGetDeviceCount(&ndev));
   CUDAERR(hipEventCreate(&start));
   CUDAERR(hipEventCreate(&stop));

   if (!ImageDims(Entrada, &DimX, &DimY, &HeadSize)) { printf("Error: File Image not BMP\n"); return -1; }

   CHECKNULL(Head =(unsigned char *)calloc(HeadSize, sizeof(unsigned char)));
   CHECKNULL(Image=(double *)calloc(DimX*DimY, sizeof(double)));

   if (!ImageLoad(Entrada, DimX, DimY, Image, Head, HeadSize)) { printf("Error: reading image\n"); return -1; }

   /* El alumnado prepara aqui los datos */

   hipEventRecord(start, 0);
     /* aqui lo que determine cada cual */
     
     /* aqui empieza el suavizado */
     for (i=1; i<=Repet; i++)
     {
        /* llamada al (los) kernel para suavizar */
     }

     /* ahora el calculo del promedio */

     /* ahora la llamada al kernel para binarizar */

     /* ahora la llamada al kernel para perfilar */

     CHECKLASTERR();
         
     /* lo que proceda para que el resultado este en el puntero Image */
     
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
   printf("\nTiempo %2.7E segundos\n", time/1000.0);

   if (!ImageSave(Salida, DimX, DimY, Image, Head, HeadSize)) { printf("Error: writing image\n"); return -1; }

   CUDAERR(hipEventDestroy(start));
   CUDAERR(hipEventDestroy(stop));

   /* Liberaciones de memoria que hagan falta */

   free(Head);
   free(Image);

   return 0;
}
