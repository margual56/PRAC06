#include "Prototipos.h"
#include "Prototipos_Alumno.h"

int main(int argc, char* argv[]) 
{
   unsigned char  *Head=NULL;
   char           *Entrada=NULL, *Salida=NULL;
   unsigned int   DimX, DimY, HeadSize, ThBlk;
   double         *Image=NULL;
   int            ndev, Repet, i;
   float          time;
   hipEvent_t    start, stop;
   
   if (argc != 5) {
      printf("Usage: %s <Image input file> <Image output file> <times> <hilos por bloque>\n", argv[0]);
      return -1;
   }

   Entrada = strdup(argv[1]);
   Salida  = strdup(argv[2]);
   Repet   = atoi(argv[3]);
   ThBlk   = atoi(argv[4]);

   CUDAERR(hipGetDeviceCount(&ndev));
   CUDAERR(hipEventCreate(&start));
   CUDAERR(hipEventCreate(&stop));

   if (!ImageDims(Entrada, &DimX, &DimY, &HeadSize)) { printf("Error: File Image not BMP\n"); return -1; }

   CHECKNULL(Head =(unsigned char *)calloc(HeadSize, sizeof(unsigned char)));
   CHECKNULL(Image=(double *)calloc(DimX*DimY, sizeof(double)));

   if (!ImageLoad(Entrada, DimX, DimY, Image, Head, HeadSize)) { printf("Error: reading image\n"); return -1; }

   /* El alumnado prepara aqui los datos */

   // Copiar los valores de GAUSS a la GPU
   double Host_GAUSS[5] = {1.0/16.0, 4.0/16.0, 6.0/16.0, 4.0/16.0, 1.0/16.0};
   double *Device_GAUSS;
   hipMalloc((void **) &Device_GAUSS, 5*sizeof(double));

   // Alocar memoria para las imágenes
   double *Device_Image, *backup, *copy;
   CUDAERR(hipMalloc((void **) &Device_Image, DimX*DimY*sizeof(double)));
   CUDAERR(hipMalloc((void **) &backup, DimX*DimY*sizeof(double)));
   CUDAERR(hipMalloc((void **) &copy, DimX*DimY*sizeof(double)));

   // Ceil the result [ (n + (p-1))/p ]
   dim3 blks (
      (DimX + (ThBlk-1)) / ThBlk,
      (DimY + (ThBlk-1)) / ThBlk
   );

   printf("Dado que hay %d threads por bloque, se ha calculado que para esta imagen hacen falta bloques de %dx%d\n",
   ThBlk, blks.x, blks.y);

   dim3 thrs (
      ThBlk,
      ThBlk
   );

   // Copiar las imágenes a la GPU

   hipEventRecord(start, 0);
     
     // Copiar datos
     CUDAERR(hipMemcpy(Device_GAUSS, Host_GAUSS, 5*sizeof(double), hipMemcpyHostToDevice)); 
     CUDAERR(hipMemcpy(Device_Image, Image, DimX*DimY*sizeof(double), hipMemcpyHostToDevice));
     //CUDAERR(hipMemcpy(backup, Device_Image, DimX*DimY, hipMemcpyDeviceToDevice));
     CUDAERR(hipMemcpy(copy, Device_Image, DimX*DimY*sizeof(double), hipMemcpyDeviceToDevice));
     
     printf("Aplicando filtro gaussiano %d veces\n", Repet);
     /* aqui empieza el suavizado */
     for (i=1; i<=Repet; i++)
     {
        /* llamada al (los) kernel para suavizar */
        kernel_Filtro1_vertical<<<blks, thrs>>>(Device_Image, copy, DimX, DimY, Device_GAUSS);
	     CHECKLASTERR();

	     kernel_Filtro1_horizontal<<<blks, thrs>>>(Device_Image, copy, DimX, DimY, Device_GAUSS);
	     CHECKLASTERR();
     }

     /* ahora el calculo del promedio */


     /* ahora la llamada al kernel para binarizar */

     /* ahora la llamada al kernel para perfilar */

     CHECKLASTERR();
         
     /* lo que proceda para que el resultado este en el puntero Image */

     CUDAERR(hipMemcpy(Image, Device_Image, DimX*DimY*sizeof(double), hipMemcpyDeviceToHost)); 
     
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
   printf("\nTiempo %2.7E segundos\n", time/1000.0);
     CHECKLASTERR();

   if (!ImageSave(Salida, DimX, DimY, Image, Head, HeadSize)) { printf("Error: writing image\n"); return -1; }

   CUDAERR(hipEventDestroy(start));
   CUDAERR(hipEventDestroy(stop));

   /* Liberaciones de memoria que hagan falta */

   free(Head);
   free(Image);
   hipFree(Device_GAUSS);
   hipFree(Device_Image);
   hipFree(copy);
   //hipFree(backup);

   return 0;
}
