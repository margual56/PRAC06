#include "Prototipos.h"
#include "Prototipos_Alumno.h"

int main(int argc, char* argv[]) 
{
   unsigned char  *Head=NULL;
   char           *Entrada=NULL, *Salida=NULL;
   unsigned int   DimX, DimY, HeadSize, ThBlk;
   double         *Image=NULL;
   int            ndev, Repet, i;
   float          time;
   hipEvent_t    start, stop;
   
   if (argc != 5) {
      printf("Usage: %s <Image input file> <Image output file> <times> <hilos por bloque>\n", argv[0]);
      return -1;
   }

   Entrada = strdup(argv[1]);
   Salida  = strdup(argv[2]);
   Repet   = atoi(argv[3]);
   ThBlk   = atoi(argv[4]);

   CUDAERR(hipGetDeviceCount(&ndev));
   CUDAERR(hipEventCreate(&start));
   CUDAERR(hipEventCreate(&stop));

   if (!ImageDims(Entrada, &DimX, &DimY, &HeadSize)) { printf("Error: File Image not BMP\n"); return -1; }

   CHECKNULL(Head =(unsigned char *)calloc(HeadSize, sizeof(unsigned char)));
   CHECKNULL(Image=(double *)calloc(DimX*DimY, sizeof(double)));

   if (!ImageLoad(Entrada, DimX, DimY, Image, Head, HeadSize)) { printf("Error: reading image\n"); return -1; }

   /* El alumnado prepara aqui los datos */

   // Copiar los valores de GAUSS a la GPU
   double Host_GAUSS[5] = {1.0/16.0, 4.0/16.0, 6.0/16.0, 4.0/16.0, 1.0/16.0};
   double *Device_GAUSS;
   hipMalloc((void **) &Device_GAUSS, 5*sizeof(double));

   // Alocar memoria para las imágenes
   double *Device_Image, *backup, *copy;
   CUDAERR(hipMalloc((void **) &Device_Image, DimX*DimY*sizeof(double)));
   CUDAERR(hipMalloc((void **) &backup, DimX*DimY*sizeof(double)));
   CUDAERR(hipMalloc((void **) &copy, DimX*DimY*sizeof(double)));

   // Ceil the result [ (n + (p-1))/p ]
   dim3 blks2D (
      (DimX + (ThBlk-1)) / ThBlk,
      (DimY + (ThBlk-1)) / ThBlk
   );

   dim3 blks1D (
      (DimX * DimY + (ThBlk-1)) / ThBlk
   );

   printf("Dado que hay %d threads por bloque, se ha calculado que para esta imagen hacen falta bloques de %dx%d\n",
   ThBlk, blks2D.x, blks2D.y);

   dim3 thrs2D (
      ThBlk,
      ThBlk
   );

   dim3 thrs1D (
      ThBlk
   );

   // Copiar las imágenes a la GPU

   hipEventRecord(start, 0);
     
     // Copiar datos
     CUDAERR(hipMemcpy(Device_GAUSS, Host_GAUSS, 5*sizeof(double), hipMemcpyHostToDevice)); 
     CUDAERR(hipMemcpy(Device_Image, Image, DimX*DimY*sizeof(double), hipMemcpyHostToDevice));
     CUDAERR(hipMemcpy(backup, Device_Image, DimX*DimY*sizeof(double), hipMemcpyDeviceToDevice));
     CUDAERR(hipMemcpy(copy, Device_Image, DimX*DimY*sizeof(double), hipMemcpyDeviceToDevice));
     
     printf("Aplicando filtro gaussiano %d veces\n", Repet);
     /* aqui empieza el suavizado */
     for (i=1; i<=Repet; i++)
     {
        /* llamada al (los) kernel para suavizar */
        kernel_Filtro1_vertical<<<blks2D, thrs2D>>>(Device_Image, copy, DimX, DimY, Device_GAUSS);
	     CHECKLASTERR();

	     kernel_Filtro1_horizontal<<<blks2D, thrs2D>>>(Device_Image, copy, DimX, DimY, Device_GAUSS);
	     CHECKLASTERR();
     }

     /* ahora el calculo del promedio */
     // TODO: Custom reduction kernel
     // kernel_Filtro2<<<blks1D, thrs1D>>>(double *IMG, const unsigned int DimX, const unsigned int DimY, const double value);

     double avg = 0;
     hipblasHandle_t handle;
     CUBLASERR(hipblasCreate(&handle));
     CUBLASERR(hipblasDasum(handle, DimX*DimY, Device_Image, 1, &avg));
     CUBLASERR(hipblasDestroy(handle));
     avg /= DimX*DimY;

     printf("El valor medio de la imagen es %.4f\n", avg);

     /* ahora la llamada al kernel para binarizar */
     kernel_Filtro3<<<blks2D, thrs2D>>>(Device_Image, DimX, DimY, avg);
     CHECKLASTERR();

     /* ahora la llamada al kernel para perfilar */
     kernel_Filtro4<<<blks2D, thrs2D>>>(Device_Image, backup, DimX, DimY, avg);

     CHECKLASTERR();
         
     /* lo que proceda para que el resultado este en el puntero Image */

     CUDAERR(hipMemcpy(Image, Device_Image, DimX*DimY*sizeof(double), hipMemcpyDeviceToHost)); 
     
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
   printf("\nTiempo %2.7E segundos\n", time/1000.0);
     CHECKLASTERR();

   if (!ImageSave(Salida, DimX, DimY, Image, Head, HeadSize)) { printf("Error: writing image\n"); return -1; }

   CUDAERR(hipEventDestroy(start));
   CUDAERR(hipEventDestroy(stop));

   /* Liberaciones de memoria que hagan falta */

   free(Head);
   free(Image);
   hipFree(Device_GAUSS);
   hipFree(Device_Image);
   hipFree(copy);
   hipFree(backup);

   return 0;
}
