#include "hip/hip_runtime.h"
#include "Prototipos_Alumno.h"

__global__ void kernel_Filtro1(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, double *GAUSS){
	int half = 2;

    //extern __shared__ double *IMG2[DimX*DimY];

    int i = blockIdx.y * blockDim.y + threadIdx.y + half;
    int j = blockIdx.x * blockDim.x + threadIdx.x + half;

    //if( i < DimY && j<DimX)
    //    IMG2[i*DimX+j] = IMG1[i*DimX+j];

    //__syncthreads();

	int f, iDx;
	double sum;

	// Filtro por columnas
    if(i < DimY-half*2){
        iDx = i*DimX;
	    //for (i = half; i<DimY-half; i++){
	    	//iDx = i*DimX;

            if(j < DimX-half*2){
	    	    //for (j = half; j<DimX-half; j++){
	    	    	sum = 0.0;
                    
	    	    	for(f = 0; f<5; f++)
	    	    		sum += IMG1[iDx+(f-half)*DimX + j]*GAUSS[f];
                    
	    	    	IMG2[iDx+j] = sum;
	    	    //}
	    //}

                    __syncthreads();

	    // Filtro por filas
	    //for (i = half; i<DimY-half; i++){
	    	//iDx = i*DimX;

	    	    //for (j = half; j<DimX-half; j++){
	    	    	sum = 0.0;
                
	    	    	for(f = 0; f<5; f++)
	    	    		sum += IMG2[iDx + j + (f-half)]*GAUSS[f];
                
	    	    	IMG1[iDx+j] = sum;
	    	    //}
            }
	    //}
    }
}