#include "hip/hip_runtime.h"
#include "Prototipos_Alumno.h"

__global__ void kernel_Filtro1(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, const int times, double GAUSS){
	int half = 2;

    int i = blockIdx.x * blockDim.x + threadIdx.x + half;
    int j = blockIdx.y * blockDim.y + threadIdx.y + half;
	int f;
	double sum;

	// Filtro por columnas
    if(i<(DimY-half)*DimX){
	    //for (i = half; i<DimY-half; i++){
	    	//iDx = i*DimX;

            if(j < DimX-half){
	    	    //for (j = half; j<DimX-half; j++){
	    	    	sum = 0.0;
                    
	    	    	for(f = 0; f<5; f++)
	    	    		sum += IMG1[i+(f-half)*DimX + j]*GAUSS[f];
                    
	    	    	IMG2[i+j] = sum;
	    	    //}
	    //}

	    // Filtro por filas
	    //for (i = half; i<DimY-half; i++){
	    	//iDx = i*DimX;

	    	    //for (j = half; j<DimX-half; j++){
	    	    	sum = 0.0;
                
	    	    	for(f = 0; f<5; f++)
	    	    		sum += IMG2[i + j + (f-half)]*GAUSS[f];
                
	    	    	IMG1[i+j] = sum;
	    	    //}
            }
	    //}
    }
}