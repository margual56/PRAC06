#include "hip/hip_runtime.h"
#include "Prototipos_Alumno.h"

__global__ void kernel_Filtro1_vertical(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, double *GAUSS){
	int half = 2;

    //extern __shared__ double *IMG2[];

    int j = blockIdx.x * blockDim.x + threadIdx.x + half;
    int i = blockIdx.y * blockDim.y + threadIdx.y + half;

    //if( i < DimY && j<DimX)
    //    IMG2[i*DimX+j] = IMG1[i*DimX+j];

    //__syncthreads();

	// Filtro por columnas
    if(i < DimY-half && j < DimX-half) {
		IMG2[i*DimX+j] = 
			IMG1[(i-2)*DimX + j] * 1.0/16.0 + 
			IMG1[(i-1)*DimX + j] * 4.0/16.0 + 
			IMG1[(i+0)*DimX + j] * 6.0/16.0 + 
			IMG1[(i+1)*DimX + j] * 4.0/16.0 + 
			IMG1[(i+2)*DimX + j] * 1.0/16.0; 
    }
}

__global__ void kernel_Filtro1_horizontal(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, double *GAUintSS){
	int half = 2;

    //extern __shared__ double *IMG2[];

    int j = blockIdx.x * blockDim.x + threadIdx.x + half;
    int i = blockIdx.y * blockDim.y + threadIdx.y + half;

    //if( i < DimY && j<DimX)
    //    IMG2[i*DimX+j] = IMG1[i*DimX+j];

    //__syncthreads();

	int iDx;

	// Filtro por columnas
    if(i < DimY-half && j < DimX-half) {
        iDx = i*DimX;
		
		IMG1[iDx+j] = 
			IMG2[iDx + (j-2)] * 1.0/16.0 + 
			IMG2[iDx + (j-1)] * 4.0/16.0 + 
			IMG2[iDx + (j+0)] * 6.0/16.0 + 
			IMG2[iDx + (j+1)] * 4.0/16.0 + 
			IMG2[iDx + (j+2)] * 1.0/16.0;

	}
}
