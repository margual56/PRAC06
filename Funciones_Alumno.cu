#include "hip/hip_runtime.h"
#include "Prototipos_Alumno.h"

__global__ void kernel_Filtro1_vertical(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, double *GAUSS)
{
	int half = 2;
    int t = blockIdx.x * blockDim.x + threadIdx.x;
	int j = t/DimX + half;
   int i = t % DimX + half;

       // int Dim_2 = blockDim.x*blockDim.y;
       //    if(i < Dim_2 && j < Dim_2) {
       //        IMG1_2[i*Dim_2+j] = IMG1[i*DimX+j];
       //      __syncthreads();
       // }

   if(i < DimY-half && j < DimX-half)
       {
               // Filtro por columnas
               IMG2[i*DimX+j] =
                       IMG1[(i-2)*DimX + j] * GAUSS[0] +
                       IMG1[(i-1)*DimX + j] * GAUSS[1] +
                       IMG1[(i+0)*DimX + j] * GAUSS[2] +
                       IMG1[(i+1)*DimX + j] * GAUSS[3] +
                       IMG1[(i+2)*DimX + j] * GAUSS[4];
   }
}

__global__ void kernel_Filtro1_horizontal(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, double *GAUSS)
{
	int half = 2;

    //extern __shared__ double *IMG2[];

    int j = blockIdx.x * blockDim.x + threadIdx.x + half;
    int i = blockIdx.y * blockDim.y + threadIdx.y + half;

    //if( i < DimY && j<DimX)
    //    IMG2[i*DimX+j] = IMG1[i*DimX+j];

    //__syncthreads();

	int iDx;

	// Filtro por columnas
    if(i < DimY-half && j < DimX-half)
	{
        iDx = i*DimX;
		
		IMG1[iDx+j] = 
			IMG2[iDx + (j-2)] * GAUSS[0] + 
			IMG2[iDx + (j-1)] * GAUSS[1] + 
			IMG2[iDx + (j+0)] * GAUSS[2] + 
			IMG2[iDx + (j+1)] * GAUSS[3] + 
			IMG2[iDx + (j+2)] * GAUSS[4];

	}
}

// double Filtro2(double *IMG, const unsigned int DimX, const unsigned int DimY, double *value)
// {
//     int j = blockIdx.x * blockDim.x + threadIdx.x + half;
//     int i = blockIdx.y * blockDim.y + threadIdx.y + half;
// 
// 	double result=0;
// 	int half = 2;
// 	unsigned int size = (DimX-half*2)*(DimY-half*2);
// 	
// 	if(i<DimY-half && j<DimX-half)
// 			result += IMG[i*DimX+j]/size;
// 	
// 	value = result;
// }

__global__ void kernel_Filtro3(double *IMG, const unsigned int DimX, const unsigned int DimY, const double value)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

	if(i<DimY && j<DimX) {
		if(IMG[i*DimX+j]<=value) 
			IMG[i*DimX+j] = 255;

		else
			IMG[i*DimX+j] = 0;
	}
}


__global__ void kernel_Filtro4(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, const double value)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
	
	double v256 = (value/256.0);

	__syncthreads();

	if(i<DimY && j<DimX)
		IMG1[i*DimX+j] = IMG1[i*DimX+j]*v256 + IMG2[i*DimX+j]*(1-v256);
}
