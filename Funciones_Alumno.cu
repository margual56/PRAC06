#include "hip/hip_runtime.h"
#include "Prototipos_Alumno.h"

__global__ void kernel_Filtro1_vertical(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, double *GAUSS){
    //extern __shared__ double IMG1_2[];

	int half = 2;
    int j = blockIdx.x * blockDim.x + threadIdx.x + half;
    int i = blockIdx.y * blockDim.y + threadIdx.y + half;

	// int Dim_2 = blockDim.x*blockDim.y;
	//    if(i < Dim_2 && j < Dim_2) {
	//        IMG1_2[i*Dim_2+j] = IMG1[i*DimX+j];
	//  	__syncthreads();
	// }

    if(i < DimY-half && j < DimX-half) {
		// Filtro por columnas
		IMG2[i*DimX+j] = 
			IMG1[(i-2)*DimX + j] * GAUSS[0] + 
			IMG1[(i-1)*DimX + j] * GAUSS[1] + 
			IMG1[(i+0)*DimX + j] * GAUSS[2] + 
			IMG1[(i+1)*DimX + j] * GAUSS[3] + 
			IMG1[(i+2)*DimX + j] * GAUSS[4]; 
    }
}

__global__ void kernel_Filtro1_horizontal(double *IMG1, double *IMG2, const unsigned int DimX, const unsigned int DimY, double *GAUSS){
	int half = 2;

    //extern __shared__ double *IMG2[];

    int j = blockIdx.x * blockDim.x + threadIdx.x + half;
    int i = blockIdx.y * blockDim.y + threadIdx.y + half;

    //if( i < DimY && j<DimX)
    //    IMG2[i*DimX+j] = IMG1[i*DimX+j];

    //__syncthreads();

	int iDx;

	// Filtro por columnas
    if(i < DimY-half && j < DimX-half) {
        iDx = i*DimX;
		
		IMG1[iDx+j] = 
			IMG2[iDx + (j-2)] * GAUSS[0] + 
			IMG2[iDx + (j-1)] * GAUSS[1] + 
			IMG2[iDx + (j+0)] * GAUSS[2] + 
			IMG2[iDx + (j+1)] * GAUSS[3] + 
			IMG2[iDx + (j+2)] * GAUSS[4];

	}
}
